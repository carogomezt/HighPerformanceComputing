#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>

using namespace std;

#define MRows 5
#define MCols 5
#define NRows 5
#define NCols 6
#define PRows 5
#define PCols 6
#define H 10
#define W 10
#define TILE_WIDTH 1

__global__ void MultTiled(float *M, float *N, float *P) {
  __shared__ int ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ int ds_N[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float PValue = 0.0;
  for (int p = 0; p < MCols / TILE_WIDTH; p++) {

    if (Row < MRows && (p * TILE_WIDTH + tx) < MCols)
      ds_M[ty][tx] = M[Row * MCols + (p * TILE_WIDTH + tx)];
    else
      ds_M[ty][tx] = 0.0;

    if (Col < NCols && (p * TILE_WIDTH + ty) < MCols)
      ds_N[ty][tx] = N[(p * TILE_WIDTH + ty) * NCols + Col];
    else
      ds_N[ty][tx] = 0.0;

    __syncthreads();

    for (int n = 0; n < TILE_WIDTH; ++n)
      PValue += ds_M[ty][n] * ds_N[n][tx];

    __syncthreads();
  }

  if (Row < PRows && Col < PCols)
    P[(Row * PCols) + Col] = PValue;
}

__global__ void multMatCUDA(float *d_a, float *d_b, float *d_c) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < PRows && col < PCols) {
    float result = 0;
    for (int k = 0; k < MCols; k++) {
      result += d_a[row * MCols + k] * d_b[k * NCols + col];
    }
    d_c[row * PCols + col] = result;
  }
}

void printMatrix(float *result, int R, int C) {
  for (int i = 0; i < R; i++) {
    for (int j = 0; j < C; j++) {
      cout << result[i * C + j] << " ";
    }
    cout << endl;
  }
}

int main() {
  clock_t start, end;
  double gpu_time_used, tiles_time_used;
  float blockSize = 1;
  float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_result, *M, *N, *P, *t_result;

  // Asignar memoria en el host
  h_a = (float *)malloc(sizeof(float) * MRows * MCols);
  h_b = (float *)malloc(sizeof(float) * NRows * NCols);
  h_c = (float *)malloc(sizeof(float) * PRows * PCols);
  h_result = (float *)malloc(sizeof(float) * PRows * PCols);
  t_result = (float *)malloc(sizeof(float) * PRows * PCols);

  // Inicializar las matrices
  for (int i = 0; i < MRows; i++) {
    for (int j = 0; j < MCols; j++) {
      h_a[i * MCols + j] = 1.0;
    }
  }
  cout << "  M1  " << endl;
  printMatrix(h_a, MRows, MCols);

  for (int i = 0; i < NRows; i++) {
    for (int j = 0; j < NCols; j++) {
      h_b[i * NCols + j] = 1.0;
    }
  }

  cout << "  M2  " << endl;
  printMatrix(h_b, NRows, NCols);
  // Asignacion de memoria en el device
  hipMalloc(&d_a, sizeof(float) * MRows * MCols);
  hipMalloc(&d_b, sizeof(float) * NRows * NCols);
  hipMalloc(&d_c, sizeof(float) * PRows * PCols);
  hipMalloc(&M, sizeof(float) * MRows * MCols);
  hipMalloc(&N, sizeof(float) * NRows * NCols);
  hipMalloc(&P, sizeof(float) * PRows * PCols);

  // Copiar los datos del host al device
  hipMemcpy(d_a, h_a, MRows * MCols * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NRows * NCols * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(M, h_a, MRows * MCols * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(N, h_b, NRows * NCols * sizeof(float), hipMemcpyHostToDevice);

  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(ceil(PCols / float(blockSize)), ceil(PRows / float(blockSize)),
               1);

  start = clock();
  multMatCUDA<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);
  hipMemcpy(h_result, d_c, PRows * PCols * sizeof(float),
             hipMemcpyDeviceToHost);
  end = clock();
  gpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
  printf("Tiempo invertido GPU = %lf s\n", gpu_time_used);

  start = clock();
  MultTiled<<<dimGrid, dimBlock>>>(M, N, P);
  hipDeviceSynchronize();
  hipMemcpy(t_result, P, PRows * PCols * sizeof(float),
             hipMemcpyDeviceToHost);
  end = clock();
  tiles_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
  printf("Tiempo invertido GPU con Tiles = %lf s\n", tiles_time_used);

  printMatrix(h_result, PRows, PCols);
  cout << "Tiles " << endl;
  printMatrix(t_result, PRows, PCols);
  // cout<<"tiles :"<<endl;
  // printMatrix(t_result);
  // if (compareTo(t_result, h_result)) {
  //   printf("Matrices Iguales");
  // } else {
  //   printf("Matrices Diferentes");
  // }
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipFree(M);
  hipFree(N);
  hipFree(P);
  free(h_a);
  free(h_b);
  free(h_c);
  free(h_result);
  free(t_result);
  return 0;
}
