#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Multiplicacion de Fila - Matriz

__global__ void multMatCUDA(double *d_a, double *d_b, double *d_c, int NRA,
                            int NCA, int NCB) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < NRA && col < NCB) {
    double result = 0;
    for (int j = 0; j < NCA; j++) {
      result += d_a[row * NCA + j] * d_b[j * NCB + col];
    }
    d_c[row * NCB + col] = result;
  }
}

void multMat(double *M_a, double *M_b, double *R_c, int NRA, int NCA, int NCB) {
  int blockSize = 32;
  double *d_a, *d_b, *d_c;

  printf("MAT A\n");
  for (int i = 0; i < NRA; i++) {
    for (int j = 0; j < NCA; j++) {
      printf("%f ", M_a[i * NCA + j]);
    }
    printf("\n");
  }

  printf("MAT B\n");
  for (int i = 0; i < NCA; i++) {
    for (int j = 0; j < NCB; j++) {
      printf("%f ", M_b[i * NCB + j]);
    }
    printf("\n");
  }

  // Asignacion de memoria en el device
  hipMalloc(&d_a, sizeof(double) * NRA * NCA);
  hipMalloc(&d_b, sizeof(double) * NCA * NCB);
  hipMalloc(&d_c, sizeof(double) * NRA * NCB);

  hipMemcpy(d_a, M_a, NRA * NCA * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, M_b, NCA * NCB * sizeof(double), hipMemcpyHostToDevice);

  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(ceil(NCB / blockSize), ceil(NRA / blockSize), 1);

  multMatCUDA<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, NRA, NCA, NCB);
  hipMemcpy(R_c, d_c, NRA * NCB * sizeof(double), hipMemcpyDeviceToHost);

  printf("MAT CUDA\n");
  for (size_t i = 0; i < NRA; i++) {
    for (size_t j = 0; j < NCB; j++) {
      printf("%f ", R_c[i * NCB + j]);
    }
    printf("\n");
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
